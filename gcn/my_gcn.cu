#include "hip/hip_runtime.h"
/*
 * @Description: load the trt engine model
 * @Author: HLG
 * @Date: 2020年6月25日21:48:46

 */
#include<fstream>
#include <cstring>
#include <NvInfer.h>
#include <hip/hip_runtime.h>
#include<NvInferPlugin.h>
#include "hip/hip_runtime.h"
#include "spdlog/spdlog.h"
#include"my_gcn.hpp"
using namespace std;
#ifndef CUDA_CHECK
#define CUDA_CHECK(callstr)                                                                    \
    {                                                                                          \
        hipError_t error_code = callstr;                                                      \
        if (error_code != hipSuccess) {                                                       \
            std::cerr << "CUDA error " << error_code << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(0);                                                                         \
        }                                                                                      \
    }
#endif
inline void* safeCudaMalloc(size_t memSize) {
    void* deviceMem;
    CUDA_CHECK(hipMalloc(&deviceMem, memSize));
    if (deviceMem == nullptr) {
        std::cerr << "Out of memory" << std::endl;
        exit(1);
    }
    return deviceMem;
}
inline int64_t volume(const nvinfer1::Dims& d)
{
    return std::accumulate(d.d, d.d + d.nbDims, 1, std::multiplies<int64_t>());
}

inline unsigned int getElementSize(nvinfer1::DataType t)
{
    switch (t)
    {
        case nvinfer1::DataType::kINT32: return 4;
        case nvinfer1::DataType::kFLOAT: return 4;
        case nvinfer1::DataType::kHALF: return 2;
        case nvinfer1::DataType::kINT8: return 1;
        default: throw std::runtime_error("Invalid DataType.");
    }
}
inline void safeCudaFree(void* deviceMem) {
    CUDA_CHECK(hipFree(deviceMem));
}

void My_trt::CopyFromHostToDevice(const std::vector<float>& input, int bindIndex) {
    CUDA_CHECK(hipMemcpy(mBinding[bindIndex], input.data(), mBindingSize[bindIndex], hipMemcpyHostToDevice));
}
void My_trt::CopyFromDeviceToHost(std::vector<float>& output, int bindIndex) {
    CUDA_CHECK(hipMemcpy(output.data(), mBinding[bindIndex], mBindingSize[bindIndex], hipMemcpyDeviceToHost));
}
My_trt::~My_trt() {
    // spdlog::info("hello!!");
    if(mContext != nullptr) {
        mContext->destroy();
        mContext = nullptr;
    }
    // spdlog::info("hello!!");
    if(mEngine !=nullptr) {
        mEngine->destroy();
        mEngine = nullptr;
    }
    // spdlog::info("hello!!");
    for(size_t i=0;i<mBinding.size();i++) {
        safeCudaFree(mBinding[i]);
    }

}
size_t My_trt::GetBindingSize(int bindIndex) const {
    return mBindingSize[bindIndex];
}
bool My_trt::DeserializeEngine(const std::string& engineFile) {
    std::ifstream in(engineFile.c_str(), std::ifstream::binary);
    if(in.is_open()) {
        spdlog::info("deserialize engine from {}",engineFile);
        auto const start_pos = in.tellg();
        in.ignore(std::numeric_limits<std::streamsize>::max());
        size_t bufCount = in.gcount();
        in.seekg(start_pos);
        std::unique_ptr<char[]> engineBuf(new char[bufCount]);
        in.read(engineBuf.get(), bufCount);
        initLibNvInferPlugins(&mLogger, "");
        mRuntime = nvinfer1::createInferRuntime(mLogger);
        mEngine = mRuntime->deserializeCudaEngine((void*)engineBuf.get(), bufCount, nullptr);
        assert(mEngine != nullptr);
        mBatchSize = mEngine->getMaxBatchSize();
        spdlog::info("max batch size of deserialized engine: {}",mEngine->getMaxBatchSize());
        // spdlog::info("DeserializeEngine!!!!");
        mRuntime->destroy();
        // spdlog::info("DeserializeEngine!!!!");
        return true;
    }
    return false;
}

void My_trt::InitEngine() {
    spdlog::info("init engine...");
    mContext = mEngine->createExecutionContext();
    assert(mContext != nullptr);

    spdlog::info("malloc device memory");
    int nbBindings = mEngine->getNbBindings();//engine.getNbBindings()是为了获取与这个engine相关的输入输出tensor的数量。//这个地方，输入+输出 总共就2个
    std::cout << "nbBingdings: " << nbBindings << std::endl;
    mBinding.resize(nbBindings);
    mBindingSize.resize(nbBindings);
    mBindingName.resize(nbBindings);
    mBindingDims.resize(nbBindings);
    mBindingDataType.resize(nbBindings);
    for(int i=0; i< nbBindings; i++) {
        nvinfer1::Dims dims = mEngine->getBindingDimensions(i);
        nvinfer1::DataType dtype = mEngine->getBindingDataType(i);
        const char* name = mEngine->getBindingName(i);
        int64_t totalSize = volume(dims) * mBatchSize * getElementSize(dtype);
        mBindingSize[i] = totalSize;
        mBindingName[i] = name;
        mBindingDims[i] = dims;
        mBindingDataType[i] = dtype;
        if(mEngine->bindingIsInput(i)) {
            spdlog::info("input: ");
        } else {
            spdlog::info("output: ");
        }
        spdlog::info("binding bindIndex: {}, name: {}, size in byte: {}",i,name,totalSize);
        spdlog::info("binding dims with {} dimemsion",dims.nbDims);
        for(int j=0;j<dims.nbDims;j++) {
            std::cout << dims.d[j] << " x ";
        }
        std::cout << "\b\b  "<< std::endl;
        mBinding[i] = safeCudaMalloc(totalSize);
        if(mEngine->bindingIsInput(i)) {
            mInputSize++;
        }
    }

}
void My_trt::DoInference(std::vector<float>& inputData, std::vector<float>& result)
{
    this->CopyFromHostToDevice(inputData, 0);//把inputData 复制到mBinding[0]
    this->Forward();
    result.resize(7); 
    this->CopyFromDeviceToHost(result,1);//把结果从mBinding[1]中取出来，放到net_output中
}
void My_trt::Forward() {
    hipEvent_t start,stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    mContext->execute(mBatchSize, &mBinding[0]);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    spdlog::info("net forward takes {} ms", elapsedTime);
}
